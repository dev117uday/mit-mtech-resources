//%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

__global__ void sort(char *s, int n, char *b)
{
    int id, i, c = 0;
    id = threadIdx.x;
    for (i = 0; i < n; i++)
    {
        if (s[i] < s[id] || s[i] == s[id] && i < id)
        {
            c++;
        }
    }
    b[c] = s[id];
}

int main(void)
{
    char s[100] = "udayyadav", b[100];
    int n, i;
    printf("Enter the string\n");
    // scanf("%s", s);

    n = strlen(s);
    char *d_s, *d_b;
    int size;

    size = sizeof(char);

    hipMalloc((void **)&d_s, n * size);
    hipMalloc((void **)&d_b, n * size);

    hipMemcpy(d_s, s, n * size, hipMemcpyHostToDevice);

    sort<<<1, n>>>(d_s, n, d_b);

    hipMemcpy(b, d_b, n * size, hipMemcpyDeviceToHost);

    printf("Output string : %s", b);

    // for (i = 0; i < n; i++)
    //     printf("%c", b[i]);

    hipFree(d_s);
    hipFree(d_b);

    return 0;
}