//%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void Add(int *a, int *b, int *c, int *d, int n)
{
    int id, i;
    id = threadIdx.x;
    for (i = 0; i < n; i++)
    {
        c[i * n + id] = a[i * n + id] + b[i * n + id];
        d[id * n + i] = a[id * n + i] + b[id * n + i];
    }
}

__global__ void AddEach(int *a, int *b, int *e, int n)
{
    int id, i;
    id = threadIdx.x;
    e[id] = a[id] + b[id];
}

int main(void)
{
    int a[100], b[100], c[100], d[100], e[100], n, i;
    int size, *d_a, *d_b, *d_c, *d_d, *d_e;

    printf("\nEnter N\n");
    // scanf("%d", &n);
    n = 3;

    printf("\nEnter MATRIX A\n");
    for (i = 0; i < n * n; i++)
    {
        // scanf("%d", &a[i]);
        a[i] = rand()%10;
        printf("%d  ", a[i] );
    }

    printf("\nEnter MATRIX B\n");
    for (i = 0; i < n * n; i++)
    {
        // scanf("%d", &b[i]);
        b[i] = rand()%10;
        printf("%d  ", b[i] );
    }

    size = sizeof(int);

    hipMalloc((void **)&d_a, n * n * size);
    hipMalloc((void **)&d_b, n * n * size);
    hipMalloc((void **)&d_c, n * n * size);
    hipMalloc((void **)&d_d, n * n * size);
    hipMalloc((void **)&d_e, n * n * size);

    hipMemcpy(d_a, a, n * n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * n * size, hipMemcpyHostToDevice);

    Add<<<1, n>>>(d_a, d_b, d_c, d_d, n);
    AddEach<<<1, n * n>>>(d_a, d_b, d_e, n);

    hipMemcpy(c, d_c, size * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(d, d_d, size * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(e, d_e, size * n * n, hipMemcpyDeviceToHost);

    printf("\nAddition of through rows \n");
    for (i = 0; i < n * n; i++)
    {
        printf("%d ", c[i]);
    }
    printf("\nAddition through cols\n");
    for (i = 0; i < n * n; i++)
    {
        printf("%d ", d[i]);
    }
    printf("\nAddition through each thread\n");
    for (i = 0; i < n * n; i++)
    {
        printf("%d ", e[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
    return 0;
}