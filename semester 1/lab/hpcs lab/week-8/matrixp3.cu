//%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 3 // Change this to the desired matrix size

__global__ void diagonalManipulation(int* matrix) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row == col) {
        matrix[row * N + col] = 0;
    } else if (row < col) {
        // Calculate the factorial of the element
        int fact = 1;
        for (int i = 2; i <= matrix[row * N + col]; i++) {
            fact *= i;
        }
        matrix[row * N + col] = fact;
    } else {
        // Calculate the sum of digits of the element
        int num = matrix[row * N + col];
        int sum = 0;
        while (num > 0) {
            sum += num % 10;
            num /= 10;
        }
        matrix[row * N + col] = sum;
    }
}

int main() {
    int matrix[N][N];

    // Initialize the matrix with some values
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i][j] = rand() % 10; // Random values between 0 and 9
        }
    }

    int* d_matrix;
    hipMalloc((void**)&d_matrix, N * N * sizeof(int));
    hipMemcpy(d_matrix, matrix, N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 dimBlock(N, N);
    dim3 dimGrid(1, 1);

    diagonalManipulation<<<dimGrid, dimBlock>>>(d_matrix);
    hipDeviceSynchronize();

    hipMemcpy(matrix, d_matrix, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Print the modified matrix
    printf("Modified Matrix:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%d\t", matrix[i][j]);
        }
        printf("\n");
    }

    hipFree(d_matrix);
    return 0;
}
