//%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>
#define Block_Size 2

__global__ void matrixmul(int *md, int *nd, int *pd, int n_wid)
{
    int a;
    int Pvalue = 0;
    int col = blockIdx.x * Block_Size + threadIdx.x;
    int row = blockIdx.y * Block_Size + threadIdx.y;

    for (a = 0; a < n_wid; a++)
    {
        Pvalue += (md[row * n_wid + a] * nd[a * n_wid + col]);
    }
    pd[row * n_wid + col] = Pvalue;
}
int main(void)
{
    int N = 4, i, j, sz;
    int A[4][4];
    int B[4][4];
    int C[4][4];
    int *d_a, *d_b, *d_c;

    printf("\ngiven matrix:\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            A[i][j] = 1;
            B[i][j] = 1;
            C[i][j] = 1;
        }
    }
    printf("\nMatrix A:\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            printf("%d ", A[i][j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            printf("%d ", B[i][j]);
        }
        printf("\n");
    }
    sz = sizeof(int) * N * N;
    int Grid_Sz;
    Grid_Sz = N / Block_Size;

    hipMalloc((void **)&d_a, sz);
    hipMalloc((void **)&d_b, sz);
    hipMalloc((void **)&d_c, sz);

    hipMemcpy(d_a, A, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, sz, hipMemcpyHostToDevice);

    dim3 blockDim(Block_Size, Block_Size);
    dim3 gridDim(Grid_Sz, Grid_Sz);

    matrixmul<<<gridDim, blockDim>>>(d_a, d_b, d_c, N);

    hipMemcpy(C, d_c, sz, hipMemcpyDeviceToHost);
    printf("\nMatrix C:\n");
    for (i = 0; i < N; i++)
    {
        for (j = 0; j < N; j++)
        {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}