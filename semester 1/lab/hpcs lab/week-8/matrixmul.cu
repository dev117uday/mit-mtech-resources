#include "hip/hip_runtime.h"
//%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void matrixmul(int *a, int *b, int *c, int m, int n, int p)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int sum = 0, j;
    for (j = 0; j < p; j++)
    {
        sum += (a[ty * n + j] * b[j * p + tx]);
    }
    c[ty * p + tx] = sum;
}

int main(void)
{
    clock_t t;
    t = clock();
    int m, n, a[100], b[100], c[100], p, i, j;
    int size, *d_a, *d_b, *d_c;

    size = sizeof(int);
    printf("Enter value of m, n, p\n");
    // scanf("%d%d%d", &m, &n, &p);
    m = 3, n = 3, p = 3;

    printf("Enter Matrix A\n");
    for (i = 0; i < m * n; i++)
    {
        // scanf("%d", &a[i]);
        a[i] = rand() % 10;
    }

    printf("Enter Matrix B\n");
    for (i = 0; i < n * p; i++)
    {
        // scanf("%d", &b[i]);
        b[i] = rand() % 10;
    }

    hipMalloc((void **)&d_a, m * n * size);
    hipMalloc((void **)&d_b, p * n * size);
    hipMalloc((void **)&d_c, m * p * size);

    hipMemcpy(d_a, a, m * n * size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * p * size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float elapsed_time_ms;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    dim3 dimGrid(1, 1);
    dim3 dimBlock(p, p);

    matrixmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, p);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("\nTime to calculate results inside GPU is: %fms\n", elapsed_time_ms);

    hipMemcpy(c, d_c, size * m * p, hipMemcpyDeviceToHost);

    printf("\nMultiplication is :\n");
    for (i = 0; i < m; i++)
    {
        printf("\n");
        for (j = 0; j < p; j++)
        {
            printf("%d  ", c[i * m + j]);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    t = clock() - t;
    double time_taken;
    time_taken = ((double)t) / CLOCKS_PER_SEC;
    printf("\nEntire program took %f seconds to execute\n", time_taken);

    return 0;
}
