//%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>

__global__ void
convert(char *s, int n)
{
    int r = 0, k = 0;
    int tid = threadIdx.x;
    int z;
    z = s[tid];
    while (z > 0)
    {
        r = z % 10;
        z = z / 10;
        k = k * 10 + r;
    }
    s[tid] = (char)k;
}

int main(void)
{
    int n, i;
    char s[100] = ":V8b";

    printf("Enter the string.\n");
    // scanf("%s\n", s);
    printf("Input string : %s\n", s);

    n = strlen(s);
    // printf("%d",n);

    int size;
    size = sizeof(char);
    char *d_s;

    hipMalloc((void **)&d_s, n * size);
    hipMemcpy(d_s, s, n * size, hipMemcpyHostToDevice);

    convert<<<1, n>>>(d_s, n);

    hipMemcpy(s, d_s, n * size, hipMemcpyDeviceToHost);

    printf("output string : %s\n", s);
    // for (i = 0; i < n; i++)
    //     printf("%c", s[i]);

    hipFree(d_s);
    return 0;
}