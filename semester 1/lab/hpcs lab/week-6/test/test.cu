#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void octal(int *a,int *b)
{
	int tid = threadIdx.x;
	long int rem[50],i=0,length=0,x=0;
	int num=a[tid];
	while(num>7)
 	{
 		rem[i]=num%8;
 		num=num/8;
 		i++;
 		length++;
 	}

	for(i=length-1;i>=0;i--)
	{
		x=x+(rem[i]*(10*i));
	}	
	b[tid]=x;

}
int main(void)
{
	int i,a[10000],b[10000],n;
	printf("Enter value of N:");
	n=20
	printf("Enter array elements of array A\n");
	for(i=0;i<n;i++)
		a[i] = rand()%100;
	int *d_a,*d_b;
	int size = sizeof(int);
	hipMalloc((void **)&d_a,size*n);
	hipMalloc((void **)&d_b,size*n);
	hipMemcpy(d_a,a,size*n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size*n,hipMemcpyHostToDevice);
	octal<<<1,n>>>(d_a,d_b);
	hipMemcpy(b,d_b,size*n,hipMemcpyDeviceToHost);
	printf("Octal array is :");
	for(i=0;i<n;i++)
	{
		printf("%d\t",b[i]);
	}
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}
