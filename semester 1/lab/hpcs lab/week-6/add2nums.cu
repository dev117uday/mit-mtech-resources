
#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a, b, c;
    int *da, *db, *dc;

    //printf("Enter a b c : ");
    // scanf("%d %d %d", &a, &b, &c);

    a = 4, b = 5;

    int intsize = sizeof(int);

    hipMalloc((void **)&da, intsize);
    hipMalloc((void **)&db, intsize);
    hipMalloc((void **)&dc, intsize);

    hipMemcpy(da, &a, intsize, hipMemcpyHostToDevice);
    hipMemcpy(db, &b, intsize, hipMemcpyHostToDevice);

    add<<<1, 1>>>(da, db, dc);

    hipMemcpy(&c, dc, intsize, hipMemcpyDeviceToHost);

    printf("Sum c : %d", c);
    
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    

}