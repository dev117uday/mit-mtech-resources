
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>

__global__ void swaparr(int *a, int size)
{
    int tid = threadIdx.x;
    if (tid < size && tid % 2 == 0)
    {
        int temp = a[tid + 1];
        a[tid + 1] = a[tid];
        a[tid] = temp;
    }
}

int main()
{

    int a[1000], n = 20;

    printf("Enter value of N:");
    scanf("%d", &n);
    printf("Enter array elements of array A\n");
    for (int i = 0; i < n; i++)
        scanf("%d", &a[i]);

    // for (int i = 0; i < n; i++)
    //     a[i] = rand() % n;

    printf("Array is :\n");
    for (int i = 0; i < n; i++)
        printf("%d ", a[i]);

    printf("\n");

    int *da;
    int intsize = sizeof(int);

    hipMalloc((void **)&da, intsize * n);
    hipMemcpy(da, a, intsize * n, hipMemcpyHostToDevice);

    swaparr<<<1, n>>>(da, n);

    hipMemcpy(a, da, intsize * n, hipMemcpyDeviceToHost);

    printf("Sum array is : \n");
    for (int i = 0; i < n; i++)
        printf("%d ", a[i]);

    printf("\n");

    hipFree(da);
}