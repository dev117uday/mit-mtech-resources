#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <math.h>

__global__ void
octal(int *a, int *b, int size)
{
    int tid = threadIdx.x;
    long int rem[50], i = 0, length = 0, x = 0;
    int num = a[tid];

    while (num != 0)
    {
        rem[i] = num % 8;
        num = num / 8;
        i++;
        length++;
    }

    for (i = length - 1; i >= 0; i--)
    {
        x = x + (rem[i] * pow(10, i));
    }
    b[tid] = x;
}

int main(int argc, char * argv[])
{
    int i, a[10000], b[10000], n;
    printf("Enter value of N:");
    n = 20;
    printf("Enter array elements of array A\n");
    for (i = 0; i < n; i++)
        a[i] = rand() % 100;
    int *d_a, *d_b;
    int size = sizeof(int);
    hipMalloc((void **)&d_a, size * n);
    hipMalloc((void **)&d_b, size * n);

    hipMemcpy(d_a, a, size * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * n, hipMemcpyHostToDevice);
    octal<<<1, n>>>(d_a, d_b, size);
    hipMemcpy(b, d_b, size * n, hipMemcpyDeviceToHost);
    printf("Octal array is :\n");
    for (i = 0; i < n; i++)
    {
        printf("%d-%d\n", a[i], b[i]);
    }
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
