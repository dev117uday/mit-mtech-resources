#include <stdio.h>
#include "hip/hip_runtime.h"
#include <stdlib.h>

__global__ void addArray(int *a, int *b, int *c, int size)
{
    int tid = threadIdx.x;
    if (tid < size)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main()
{

    int a[1000], b[1000], c[1000], n = 20;

    // printf("Enter value of N:");
    // scanf("%d", &n);
    // printf("Enter array elements of array A\n");
    // for (i = 0; i < n; i++)
    //     scanf("%d", &a[i]);

    // printf("Enter array elements of array B\n");
    // for (i = 0; i < n; i++)
    //     scanf("%d", &b[i]);

    for (int i = 0; i < n; i++)
        a[i] = rand()%n;

    for (int i = 0; i < n; i++)
        b[i] = rand()%n;

    int *da, *db, *dc;
    int intsize = sizeof(int);

    hipMalloc((void **)&da, intsize*n);
    hipMalloc((void **)&db, intsize*n);
    hipMalloc((void **)&dc, intsize*n);

    hipMemcpy(da, a, intsize * n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, intsize * n, hipMemcpyHostToDevice);

    addArray<<<1, n>>>(da, db, dc, n);

    hipMemcpy(c, dc, intsize * n, hipMemcpyDeviceToHost);

    printf("Sum array is :");
    for (int i = 0; i < n; i++)
        printf("%d %d %d\n", a[i], b[i], c[i]);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}